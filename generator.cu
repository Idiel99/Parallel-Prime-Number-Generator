
#include <hip/hip_runtime.h>
#include <emmintrin.h>
#include <sys/time.h>
#include <stdio.h>

const long N = 1000000; 
struct timeval start, end;


void starttime() {
  gettimeofday( &start, 0 );
}

void endtime(const char* c) {
   gettimeofday( &end, 0 );
   double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
   printf("%s: %f ms\n", c, elapsed); 
}

void init(const char* c) {
  printf("***************** %s **********************\n", c); 
  printf("Running %s...\n", c);
  starttime();
}

void finish(int a, long N, const char* c) {
	endtime(c);
	printf("Done.\n");
	printf("\nThere are %ld Prime numbers between 1 and %ld.", a, N);
	printf("***************************************************\n");
}

int normal(int a, long N)
{
    long low = 2, high = N, i, check;
	// printf("Prime numbers between 1 and %d are: ",high);
	while (low < high)
	{
		check = 0;
		for(i = 2; i <= low/2; ++i)
		{
		if(low % i == 0)
		{
			check = 1;
			break;
		}
		}
		if (check == 0)
			++a;
		//printf("%d ", low);
		++low;
   }
   return a;
}                                                                                                                                                                                                       

// GPU function to countprime numbers
// Every thread on every core runs this function
__global__ void gpu_prime(int* a, long N) {
   // One element per thread on each core
   // blockIdx.x = Core #
   // blockDim.x = Threads per core
   // threadIdx.x = Thread #
   // The formula below makes sure the value of element 
   // is different on every thread on every core
   long element = blockIdx.x*blockDim.x + threadIdx.x;
   
   // If there is not an event split, some threads will be 
   // out of bounds
   // We just let those do nothing
   // The rest count the prime numbers 
   
	if (element <= N && element >= 2) {
		int check = 0;	
		for(int i = 2; i <= element/2; ++i) {
        		if(element  % i == 0) {
        		check = 1;
        		break;
        		}
        	}
		if (check == 0){
	                atomicAdd(a,1);	
		}
	}
}

void gpu(int* a, long N) {
   int threadsPerCore = 512; // This can vary, up to 1024
   long numCores = N / threadsPerCore + 1; 

  
   // Memory must be on the graphics card 
   int* gpuA;
   hipMalloc(&gpuA, sizeof(int)); // Allocate enough memory on the GPU
   
   hipMemcpy(gpuA, a, sizeof(int), hipMemcpyHostToDevice); 
   gpu_prime<<<numCores, threadsPerCore>>>(gpuA, N);
   hipMemcpy(a, gpuA, sizeof(int), hipMemcpyDeviceToHost); 
   hipFree(&gpuA); // Free the memory on the GPU
}
                                                                                                                                                                                               
 

int main()                                                                                                                                                                                  
{

	int a = 1;
	
	// Test 1: Sequential For Loop
	init ("Normal");
	a = normal(a, N); 
	finish(a, N, "Normal"); 
	// Test 2: GPU
	a = 1;
	init("GPU");
	gpu(&a, N);  
	finish(a, N, "GPU");
  
	return 0;
}

